#include "hip/hip_runtime.h"
/*
 * main.cu
 * 3-dimensional convection with time-independent velocty vector field using
 * CUDA C/C++ implementing finite difference
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160723
*/
#include <functional>

#include "./commonlib/tex_anim2d.h"
#include "./commonlib/errors.h"
#include "./commonlib/finitediff.h"
#include "./commonlib/sharedmem.h"

#include "./physlib/convect.h"
#include "./physlib/R3grid.h"
#include "./physlib/dev_R3grid.h"

#include "math.h" // CUDA C/C++ math.h

#define GL_GLEXT_PROTOTYPES // needed for identifier glGenBuffer, glBindBuffer, glBufferData, glDeleteBuffers

#include <GL/glut.h>

#include <hip/hip_runtime.h>

#include <cuda_gl_interop.h>


const float Deltat[1] { 0.00001f } ; 

// physics
const int W { 680 } ;
const int H { 680 } ;
const int DEPTH { 320 } ;

dim3 dev_L3 { static_cast<unsigned int>(W), 
				static_cast<unsigned int>(H),
				static_cast<unsigned int>(DEPTH) };
				
dev_Grid3d dev_grid3d( dev_L3 );		


// graphics + physics

const dim3 M_i { 2, 2, 2 };

const int iters_per_render { 5 } ;


GPUAnim2dTex animtexmap( W, H );
GPUAnim2dTex* texptr = &animtexmap;


// struct DataBlock for recording, benchmarking events, iterations
struct DataBlock {
	hipEvent_t  start, stop;
	float        totalTime;
	float        frames; 
};

DataBlock databenchmarks ; 



void make_render(int iters_per_render ) {
	uchar4* dev_out=0; 
	
	HANDLE_ERROR(
		hipGraphicsMapResources(1, &(texptr->cuda_pixbufferObj_resource), 0 ) );
	
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void **)&dev_out, NULL, 
			texptr->cuda_pixbufferObj_resource ) 
	);
	
	dim3 grids( (dev_L3.x+M_i.x-1)/M_i.x,(dev_L3.y+M_i.y-1)/M_i.y,(dev_L3.z+M_i.z-1)/M_i.z) ;
	
	hipEventRecord( databenchmarks.start, 0 );
	
	for (int i = 0 ; i < iters_per_render; ++i ) {
		convect_fd_naive_sh<<<grids,M_i>>>(
			dev_grid3d.dev_rho, dev_grid3d.dev_u ); 

//		convect_sh<<<grids,M_i>>>( dev_grid3d.dev_rho, dev_grid3d.dev_u ) ;

//		float_to_color3d<<<grids,M_i>>>(dev_out, dev_grid3d.dev_rho ) ;


//		float_to_char<<<grids,M_i>>>(dev_out, dev_grid3d.dev_rho ) ;

	}
	//float_to_color3d<<<grids,M_i>>>(dev_out, dev_grid3d.dev_rho ) ;

	float_to_char<<<grids,M_i>>>(dev_out, dev_grid3d.dev_rho ) ;


	// Recording time for rough benchmarking, only
	hipEventRecord( databenchmarks.stop, 0 );
	hipEventSynchronize( databenchmarks.stop );

	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, databenchmarks.start, databenchmarks.stop );

	databenchmarks.totalTime += elapsedTime;
	++databenchmarks.frames;

//	printf("Iteration complete : ticks %d \n ", ticks );
	printf("Average Time per frame: %3.1f ms \n", databenchmarks.totalTime/databenchmarks.frames );
	// END of Recording time for rough benchmarking, only, END


	HANDLE_ERROR(
		hipGraphicsUnmapResources( 1, &texptr->cuda_pixbufferObj_resource, 0 ));


	char title[128];
	sprintf(title, "mass density Visualizer - Iterations=%4d, ", iterationCount );
	
	glutSetWindowTitle(title);
}; 

std::function<void()> render = std::bind( make_render, iters_per_render ) ;



std::function<void()> draw_texture = std::bind( make_draw_texture, W,H) ;

void display() {
	render();
	draw_texture()   ;


	glutSwapBuffers();
}




int main(int argc, char** argv) {
	// physics
	constexpr float rho_0 { 0.956 };
	constexpr std::array<int,3> LdS { W, H, DEPTH} ; 
	constexpr std::array<float,3> ldS { 1.f, 1.f, 1.f };

	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL( dev_Deltat), Deltat, sizeof(float)*1,0,hipMemcpyHostToDevice) );

	const int Ld_to_const[3] { LdS[0], LdS[1], LdS[2] };
	
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL( dev_Ld), Ld_to_const, sizeof(int)*3,0,hipMemcpyHostToDevice) );


	// set radius for shared memory "tiling", i.e. for the "halo" cells
	const int radius[1] { 2 };
	HANDLE_ERROR(
		hipMemcpyToSymbol(HIP_SYMBOL( sharedmem::RAD), radius, sizeof(int)*1,0,hipMemcpyHostToDevice) );


	Grid3d grid3d( LdS, ldS);

	const float hds[3] { grid3d.hd[0], grid3d.hd[1], grid3d.hd[2] };

	set2DerivativeParameters(hds );

	// data benchmarking, cuda timing of events
	databenchmarks.totalTime = 0;
	databenchmarks.frames = 0 ;

	hipEventCreate( &databenchmarks.start );
	hipEventCreate( &databenchmarks.stop );
	


	// graphics setup
	
	texptr->initGLUT(&argc,argv) ;
	 
	glutKeyboardFunc( keyboard_func );
	glutMouseFunc( mouse_func );
	glutIdleFunc( idle );
	glutDisplayFunc( display) ;
	texptr->initPixelBuffer();
		

	// initial conditions

	for (int k=0; k<(grid3d.Ld[2]); ++k) {
		for (int j=0; j<(grid3d.Ld[1]); ++j) {
			for (int i=0;i<(grid3d.Ld[0]); ++i) {

				grid3d.u[ grid3d.flatten(i,j,k) ].x = 25.0;  // meters/second
				grid3d.u[ grid3d.flatten(i,j,k) ].y = 25.0;  // meters/second
				grid3d.u[ grid3d.flatten(i,j,k) ].z = 12.0;  // meters/second

			}
		}
	}

	std::array<int,3> ix_in { 0, 0, 0 };
	std::array<float,3> b_0 { 0.25f*grid3d.ld[0], 0.25f*grid3d.ld[1], 0.5f*grid3d.ld[2]  };
	
	for (int k=0; k<(grid3d.Ld[2]); ++k) {
		for (int j=0; j<(grid3d.Ld[1]); ++j) {
			for (int i=0; i<(grid3d.Ld[0]); ++i) {
				ix_in[0] = i;
				ix_in[1] = j;
				ix_in[2] = k;
				grid3d.rho[ grid3d.flatten(i,j,k) ] = 
						gaussian3d( rho_0, 0.05, b_0,grid3d.gridpt_to_space(ix_in));

			}
		}
	}


	HANDLE_ERROR(
		hipMemcpy( dev_grid3d.dev_rho, grid3d.rho, grid3d.NFLAT()*sizeof(float), hipMemcpyHostToDevice)
		);

	HANDLE_ERROR(
		hipMemcpy( dev_grid3d.dev_u, grid3d.u, grid3d.NFLAT()*sizeof(float3), hipMemcpyHostToDevice)
		);


	glutMainLoop();

	HANDLE_ERROR(
		hipFree( dev_grid3d.dev_rho )  );

	HANDLE_ERROR(
		hipFree( dev_grid3d.dev_u ) );
		
	texptr->exitfunc(); 
}
	
