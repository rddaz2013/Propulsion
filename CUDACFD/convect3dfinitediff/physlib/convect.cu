#include "hip/hip_runtime.h"
/* convect.cu
 * 3-dim. convection by finite difference with shared memory
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160720
 */
#include "convect.h"

__constant__ float dev_Deltat[1] ;

// convect_fd_naive_sh - convection with finite difference and naive shared memory scheme
__global__ void convect_fd_naive_sh( float* dev_rho, float3* dev_u ) {

	const int NU = 2;
	
	// map from threadIdx/blockIdx to x grid position
	int k_x = threadIdx.x + blockIdx.x * blockDim.x;
	int k_y = threadIdx.y + blockIdx.y * blockDim.y;
	int k_z = threadIdx.z + blockIdx.z * blockDim.z;
	
	int k = k_x + k_y*blockDim.x*gridDim.x + k_z*blockDim.x*gridDim.x*blockDim.y*gridDim.y ;

	__shared__ int3 stencilindicesplus[NU] ;
	__shared__ int3 stencilindicesminus[NU] ;

	for (int nu = 0; nu < NU; ++nu ) {
		stencilindicesplus[  nu ].x = k + (nu + 1) ; 
		stencilindicesminus[ nu ].x = k - (nu + 1) ; 
		stencilindicesplus[  nu ].y = k + (nu + 1)*dev_Ld[0] ; 
		stencilindicesminus[ nu ].y = k - (nu + 1)*dev_Ld[0] ; 
		stencilindicesplus[  nu ].z = k + (nu + 1)*dev_Ld[0]*dev_Ld[1] ; 
		stencilindicesminus[ nu ].z = k - (nu + 1)*dev_Ld[0]*dev_Ld[1] ; 
	}

	int XI = 0;

	// check boundary conditions
	for (int nu = 0; nu < NU; ++nu) {
		if (k_x == nu ) {
			XI = NU - nu ;
			for (int xi = 0; xi < XI; ++xi ) {
				stencilindicesminus[ NU - 1 - xi ].x += XI- xi ;  
			}
		}
	
		if (k_y == nu ) {
			XI = NU - nu ;
			for (int xi = 0; xi < XI; ++xi) { 
				stencilindicesminus[ NU - 1 - xi ].y += (XI - xi)*dev_Ld[0] ;
			}
		}
		
		if (k_z == nu) {
			XI = NU - nu ;
			for (int xi = 0 ; xi < XI; ++xi ) {
				stencilindicesminus[ NU - 1 - xi ].z += (XI - xi)*dev_Ld[0]*dev_Ld[1] ;  
			}
		}
	
		if (k_x == (dev_Ld[0] - (nu + 1) ) ) {
			XI = NU - nu ;
			for (int xi = 0; xi < XI; ++xi ) {
				stencilindicesplus[ NU - 1 - xi].x -= XI-xi ;
			}
		}
		
		if (k_y == (dev_Ld[1] - (nu + 1) ) ) {
			XI = NU - nu ;
			for (int xi = 0; xi < XI; ++xi ) {
				stencilindicesplus[ NU - 1 - xi].y -= (XI-xi)*dev_Ld[0] ;
			}
		}
		
		if (k_z == (dev_Ld[2] - (nu + 1) ) ) {
			XI = NU - nu ;
			for (int xi = 0; xi < XI; ++xi ) {
				stencilindicesplus[ NU - 1 - xi].z -= (XI-xi)*dev_Ld[0]*dev_Ld[1] ;
			}
		}
		
	}
	
//	__syncthreads();
		
	__shared__ float3 stencil[NU][2] ; 
	
	for (int nu = 0 ; nu < NU; ++nu ) {
		stencil[nu][0].x = dev_rho[stencilindicesminus[nu].x]*dev_u[stencilindicesminus[nu].x].x  ;
		stencil[nu][1].x = dev_rho[stencilindicesplus[nu].x]*dev_u[stencilindicesplus[nu].x].x  ;
		stencil[nu][0].y = dev_rho[stencilindicesminus[nu].y]*dev_u[stencilindicesminus[nu].y].y  ;
		stencil[nu][1].y = dev_rho[stencilindicesplus[nu].y]*dev_u[stencilindicesplus[nu].y].y  ;
		stencil[nu][0].z = dev_rho[stencilindicesminus[nu].z]*dev_u[stencilindicesminus[nu].z].z  ;
		stencil[nu][1].z = dev_rho[stencilindicesplus[nu].z]*dev_u[stencilindicesplus[nu].z].z  ;
	}	
	
	float div_value { dev_div2( stencil ) } ;
	
	__syncthreads();
	
	dev_rho[k] +=  dev_Deltat[0] * (-1.f) * div_value ;		
			
//	__syncthreads();		
			
}

__global__ void convect_sh( float* dev_rho, float3* dev_u ) {
	const int k_x = threadIdx.x + blockIdx.x * blockDim.x; 
	const int k_y = threadIdx.y + blockIdx.y * blockDim.y; 
	const int k_z = threadIdx.z + blockIdx.z * blockDim.z; 
	
	const int k = k_x + k_y*dev_Ld[0]+k_z*dev_Ld[0]*dev_Ld[1]   ;
	
	float div_value = sharedmem::dev_div2( dev_rho, dev_u) ;
	
	dev_rho[k] += dev_Deltat[0] * (1.f) * div_value ;
}





